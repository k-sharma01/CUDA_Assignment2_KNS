/*
* Kirin Sharma
* CS-425 Advanced Architecture
* CUDA Assignment 2
*
* This program creates and runs a CUDA kernel to multiply a matrix and a vector and store to a result vector. Based on how 
* the matrix and vector are initialized, the resulting vector should contain 10 of the same value:
*  the sum of squares 1-10 which should come out to 385
*/

#include <hip/hip_runtime.h>
#include <iostream>

#include <stdlib.h>

using namespace std;

// CUDA kernel function to multiply the matrix and the vector and store to a result vector
__global__ void multiply(int *matrix, int *vector, int* result, int size)
{
    // Used to identify which thread should work on which data and initialize shared memory
    int row = blockIdx.x;
    int col = threadIdx.x;
    __shared__ int partialSum[16];

    // Have the current thread compute its partial sum and store it to shared memory, then sync threads in the block (only use threads 1-10, rest are 0)
    if(col < size) {
        partialSum[col] = matrix[row * size + col] * vector[col];
    } else {
        partialSum[col] = 0;
    }
    __syncthreads();

    // Compute total sum in parallel from the shared memory
    for(int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if(col < stride){
            partialSum[col] += partialSum[col + stride];
        }
        __syncthreads();
    }

    // Store the final result to the result matrix
    if(col == 0) {
        result[row] = partialSum[0];
    }
}

// Helper function to print a matrix
void printMatrix(int* matrix, int size) {
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            cout << matrix[size * i + j] << "   ";
        }
        cout << "\n";
    }
}

// Helper function to print a vector
void printVector(int* vector, int size) {
    for(int i = 0; i < size; i++) {
        cout << vector[i] << "   ";
    }
}

int main()
{
    // Allocate unified memory for the (flattened) 2-D matrix and vectors
	int size = 10;
	int *matrix;
	int *vector;
    int *result;
    hipMallocManaged(&matrix, size * size * sizeof(int));
    hipMallocManaged(&vector, size * sizeof(int));
    hipMallocManaged(&result, size * sizeof(int));

    // Initialize the matrix and vector
    for(int i = 0; i < size; i++) {
        vector[i] = i + 1;
        for(int j = 0; j < size; j++) {
            matrix[i * size + j] = j + 1;
        }
    }

    // Print the starting vector
    cout << "Starting Vector:\n";
    printVector(vector, size);
    cout << "\n\n";

    // Print the starting matrix
    cout << "Starting Matrix:\n";
    printMatrix(matrix, size);
    cout << "\n\n";

    // Specify 10 blocks with 16 threads each (power of 2 even though only 10 really used per block)
    int numBlocks = 10;
    int threadsPerBlock = 16;

    // Launch the cuda kernel
    multiply<<<numBlocks, threadsPerBlock>>>(matrix, vector, result, size);
    hipDeviceSynchronize();

    // Print the result vector
    cout << "Result Vector:\n";
    printVector(result, size);

    // Free cuda memory
    hipFree(matrix);
    hipFree(vector);
    hipFree(result);

    return 0;
    
} // end main
